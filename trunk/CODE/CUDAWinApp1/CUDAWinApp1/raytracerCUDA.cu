#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "Escena.h"
#include "tipos.h"
#include "math.h"

#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <math_functions.h>
#include <cutil_math.h>


/////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////
/////////////////          DECLARACIONES           //////////////////////
/////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////


//Declaraci�n de las texturas que contendr�n la estructura de la aplicaci�n

texture<float4, 1, hipReadModeElementType> textura_triangulos;//Objetos de la escena
texture<float4, 1, hipReadModeElementType> textura_normales;//Normales de la escena
texture<float4, 1, hipReadModeElementType> textura_luces;//Luces
texture<int, 1, hipReadModeElementType> textura_voxels;//Lista de indices de la lista de objetos
texture<int, 1, hipReadModeElementType> textura_listasGrid;//Lista de elementos que se encuentran dentro de un voxel particular.
texture<float4, 1, hipReadModeElementType> textura_rayos;//Rayos a trazar.
texture<float4, 1, hipReadModeElementType> textura_materiales;//Materiales para todos los objetos de la escena.


//VARIABLES DEFINIDAS A NIVEL DE MEMORIA CONSTANTE EN LA GPU
__constant__ Configuracion configuracion_gpu;//Par�metros de configuraci�n
__constant__ float3 ojo;
__constant__ float3 dx;
__constant__ float3 dy;
__constant__ float3 ini;
__constant__ float cant_luces;
__constant__ float3 dimension_grilla;
__constant__ BoundingBox bounding_box;
__constant__ float3 tam_voxel;
__constant__ float3 tam_grilla;


//Variables utilizadas...
float3* d_color;
float4* d_listaRayos;


/////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////
/////////////////       FUNCIONES EN GPU           //////////////////////
/////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////

//#define OLD_ALGORITHM

__device__ bool intersecar(Rayo r, float &distancia, float3 & normal, int ind_obj, bool calc_normal){
	float3 v1 = make_float3(tex1Dfetch(textura_triangulos, 3 * ind_obj));
	float3 v2 = make_float3(tex1Dfetch(textura_triangulos, 3 * ind_obj + 1));
	float3 v3 = make_float3(tex1Dfetch(textura_triangulos, 3 * ind_obj + 2));
	
#ifndef OLD_ALGORITHM
	float3 lado1 = v2-v1;
	float3 lado2 = v3-v1;

	float3 p= cross(r.dir,lado2);
	float determinante = dot(lado1, p);

	if(determinante > -0.001f && determinante < 0.001f)
		return false;

	float inv_det = 1.0f/determinante;
	float3 t = ( r.origen-v1 );

	float u = dot(t,p)* inv_det;
	if(u<0.0f || u>1.0f)
		return false;

	float3 q = cross(t, lado1);

	float v = dot (r.dir, q) * inv_det;
	if(v<0.0f || u+v>1.0f)
		return false;

	float dist = dot(lado2, q)*inv_det;
	if(dist < 0)
		return false;
	
	if(dist>distancia)
		return false;

	distancia = dist;
	if(calc_normal){
		float s = 1-u-v;
		float3 normal_v1 = make_float3(tex1Dfetch(textura_normales, 3 * ind_obj));
		float3 normal_v2 = make_float3(tex1Dfetch(textura_normales, 3 * ind_obj + 1));
		float3 normal_v3 = make_float3(tex1Dfetch(textura_normales, 3 * ind_obj + 2));
		normal = (normal_v1 * s + normal_v2 * u + normal_v3 * v);
	}
	return true;
#else	

	float3 primero = v2 - v1;
	float3 segundo = v3 - v1;
	normal = normalize(cross(primero,segundo));
		
	float d = -dot(normal, v1);
	float t = -( dot(r.origen, normal) + d ) / ( dot(normal,r.dir) );
	if(t < 0.001){
		return false;
	}
	float3 I = r.origen + r.dir * t;
	if(distancia < t){
		return false;
	}
	float s1 = dot( cross(primero,(I - v1)), normal );
	float s2 = dot( cross((v3 - v2),(I - v2)), normal );
	float s3 = dot( cross((-segundo),(I - v3)), normal );
	if((s1 >= 0 && s2 >= 0 && s3 >= 0) || (s1 <= 0 && s2 <= 0 && s3 <= 0)){
		distancia = t;
		if(calc_normal){
			float3 normal_v1 = make_float3(tex1Dfetch(textura_normales, 3 * ind_obj));
			float3 normal_v2 = make_float3(tex1Dfetch(textura_normales, 3 * ind_obj + 1));
			float3 normal_v3 = make_float3(tex1Dfetch(textura_normales, 3 * ind_obj + 2));
			normal = normal_v1 * s2 + normal_v2 * s3 + normal_v3 * s1;
		}
		return true;
	}
	return false;
#endif
}


__device__ float3 coordMundoACoordGrid(float3 coordMundo){
	float3 retorno;
	if(bounding_box.minimum.x==bounding_box.maximum.x){
		retorno.x = 0.f;
	}else{
		retorno.x =(int)(((coordMundo.x-bounding_box.minimum.x)/(bounding_box.maximum.x-bounding_box.minimum.x))*(dimension_grilla.x));
	}

	if(bounding_box.minimum.y==bounding_box.maximum.y){
		retorno.y = 0.f;
	}else{
		retorno.y =(int)(((coordMundo.y-bounding_box.minimum.y)/(bounding_box.maximum.y-bounding_box.minimum.y))*(dimension_grilla.y));
	}

	if(bounding_box.minimum.z==bounding_box.maximum.z){
		retorno.z = 0.f;
	}else{
		retorno.z =(int)(((coordMundo.z-bounding_box.minimum.z)/(bounding_box.maximum.z-bounding_box.minimum.z))*(dimension_grilla.z));
	}

	if(retorno.x==dimension_grilla.x){
		retorno.x = (int)(dimension_grilla.x-1);
	}
	if(retorno.y==dimension_grilla.y){
		retorno.y = (int)(dimension_grilla.y-1);
	}
	if(retorno.z==dimension_grilla.z){
		retorno.z = (int)(dimension_grilla.z-1);
	}
	return retorno;
}


__device__ float3 coordGridACoordMundo(float3 voxelActual, float3* voxelMundo){
	*voxelMundo = bounding_box.minimum + voxelActual*tam_voxel;
	return *voxelMundo;
}

__device__ bool hallarPuntoEntradaGrilla(Rayo r, float3* puntoEntrada){
	if(r.origen.x >= bounding_box.minimum.x && r.origen.y >= bounding_box.minimum.y &&
	   r.origen.z >= bounding_box.minimum.z && r.origen.x <= bounding_box.maximum.x &&
	   r.origen.y <= bounding_box.maximum.y && r.origen.z <= bounding_box.maximum.z){
		   (*puntoEntrada) = make_float3(r.origen.x, r.origen.y, r.origen.z);
		   return true;
	}

	float min= -configuracion_gpu.INFINITO;
	float max = configuracion_gpu.INFINITO;

	float3 resta = tam_grilla;
	float3 centro = resta + bounding_box.minimum;
	float3 p = centro - r.origen;

	//Plano alineado con X
	float e = p.x;
	float f = r.dir.x;

	float h = resta.x;
	if(abs(f)>=configuracion_gpu.ZERO){
		float t1 = (e + h)/f;
		float t2 = (e - h)/f;
		if(t1>t2) {
			float aux = t1;
			t1 = t2;
			t2 = aux;
		}
		min = (t1>min)?t1:min;
		max = (t2<max)?t2:max;
		if(min>max) return false;
		if(max<0) return false;
	}
	else if(-e-h>0 || -e+h<0) return false;

	//Plano alineado con Y
	e = p.y;
	f = r.dir.y;
	h = resta.y;
	if(abs(f)>=configuracion_gpu.ZERO){
		float t1 = (e + h)/f;
		float t2 = (e - h)/f;
		if(t1>t2) {
			float aux = t1;
			t1 = t2;
			t2 = aux;
		}
		min = (t1>min)?t1:min;
		max = (t2<max)?t2:max;
		if(min>max) return false;
		if(max<0) return false;
	}
	else if(-e-h>0 || -e+h<0) return false;

	//Plano alineado con Z
	e = p.z;
	f = r.dir.z;
	h = resta.z;
	if(abs(f)>=configuracion_gpu.ZERO){
		float t1 = (e + h)/f;
		float t2 = (e - h)/f;
		if(t1>t2) {
			float aux = t1;
			t1 = t2;
			t2 = aux;
		}
		min = (t1>min)?t1:min;
		max = (t2<max)?t2:max;
		if(min>max) return false;
		if(max<0) return false;
	}
	else if(-e-h>0 || -e+h<0) return false;
	if(min>0){
		*puntoEntrada = r.origen + r.dir * min;
		return true;
	}
	*puntoEntrada = r.origen + r.dir * max;
	return true;

}



__device__ bool intersecarObjetosGrilla(int comienzoLista, Rayo r, float& distancia, float3& normal, int& indiceObjeto, bool calc_normal){
	
	if (comienzoLista == -1) 
		return false;

	bool intersecaron = false;
	indiceObjeto = comienzoLista;
	float3 normalAux;

	int valor = tex1Dfetch(textura_listasGrid, comienzoLista);
	while(valor != -1){
		bool choco = intersecar(r, distancia, normalAux, valor, calc_normal);
		if(choco){
			indiceObjeto = valor;
			if(calc_normal)
				normal = normalAux;
			intersecaron= true;
		}
		comienzoLista++;
		valor = tex1Dfetch(textura_listasGrid, comienzoLista);
	}
	return intersecaron;
}

//Calcula los vectores necesarios para la recorrida de los voxels
__device__ void calcularInicioGrilla(float3 dir, float3 curPos, float3& incre, float3& tMin){
	
	float3 voxelSize = tam_voxel;

	incre.x = abs(dir.x)> configuracion_gpu.ZERO ? (voxelSize.x)/dir.x:0;
	incre.y = abs(dir.y)> configuracion_gpu.ZERO ? (voxelSize.y)/dir.y:0;
	incre.z = abs(dir.z)> configuracion_gpu.ZERO ? (voxelSize.z)/dir.z:0;

	if(incre.x == 0)
		tMin.x = configuracion_gpu.INFINITO;
	if(incre.y == 0)
		tMin.y = configuracion_gpu.INFINITO;
	if(incre.z == 0)
		tMin.z = configuracion_gpu.INFINITO;

	float3 voxelActual;
	voxelActual = coordMundoACoordGrid(curPos);

	if(dir.x>0)
		voxelActual.x+=1.0f;
	if(dir.y>0)
		voxelActual.y+=1.0f;
	if(dir.z>0)
		voxelActual.z+=1.0f;

	float3 voxelMundo;
	coordGridACoordMundo(voxelActual,&voxelMundo);

	if(incre.x != 0)
		tMin.x = (voxelMundo.x - curPos.x)/dir.x;
	if(incre.y != 0)
		tMin.y = (voxelMundo.y - curPos.y)/dir.y;
	if(incre.z != 0)
		tMin.z = (voxelMundo.z - curPos.z)/dir.z;

}


//Devuelve false si salio de la grilla y me modifica el voxel actual en el caso de que se 
//pueda avanzar en la grilla
__device__ bool siguienteVoxel(float3 &tMin, float increX, float increY, float increZ,  float3& voxelActual){
	float tMinx = abs(tMin.x);
	float tMiny = abs(tMin.y);
	float tMinz = abs(tMin.z);
	if(tMinx<tMiny){
		if(tMinx<tMinz){ //X min
			tMin.x+=abs(increX);
			voxelActual.x+=increX>0?1.0f:-1.0f;
		}
		else{			 //Z min
			tMin.z+=abs(increZ);
			voxelActual.z+=increZ>0?1.0f:-1.0f;
		}
	}else{
		if(tMiny<tMinz){ //Y min
			tMin.y+=abs(increY);
			voxelActual.y+=increY>0?1.0f:-1.0f;
		}
		else{			 //Z min
			tMin.z+=abs(increZ);
			voxelActual.z+=increZ>0?1.0f:-1.0f;
		}
	}

	if(voxelActual.x*voxelActual.x>=dimension_grilla.x*dimension_grilla.x||
		voxelActual.y*voxelActual.y>=dimension_grilla.y*dimension_grilla.y||
		voxelActual.z*voxelActual.z>=dimension_grilla.z*dimension_grilla.z||
		voxelActual.x<0||voxelActual.y<0||voxelActual.z<0){
		return false;
	}
	return true;

}

////<<<<<<<<<<<<< HALLAR RAYOS >>>>>>>>>>>>>////
__global__ void hallarColor(float3* retorno){

		int indiceR = configuracion_gpu.resolucion.x*(blockIdx.y*blockDim.y+threadIdx.y)+(blockIdx.x*blockDim.x+threadIdx.x);

		//Rayo actual que me permite ejecutar la iteraci�n.
		Rayo rayoactual;
		rayoactual.dir = make_float3(tex1Dfetch(textura_rayos, indiceR));
		rayoactual.origen = ojo;
	
		int nivel = 0;
		bool salir = false;
		float3 voxelActual;
		float3 puntoEntrada;

		//Hallo el punto de entrada a la grilla y calculo el voxel que corresponde.
		bool entra = hallarPuntoEntradaGrilla(rayoactual, &puntoEntrada);
		voxelActual = coordMundoACoordGrid(puntoEntrada);
		
		float3 color = make_float3(0,0,0);
		float3 colorAcumulado = make_float3(0,0,0);
		
		//TODO ADD
		float multiplicador = 1.0;
		//float3 normal;


		// Para la refracci�n
		bool adentro = false;

		float3 normal = make_float3(0,0,0);
		while((nivel < configuracion_gpu.profundidad_recursion) && !salir && entra){
			
			bool salirGrilla = false;
			
			//Extra�amente si no inicializo esto no funciona... NO COMPILA!!! cuack
			float3 tMin = make_float3(0,0,0);
			float3 incre = make_float3(0,0,0);

			//Calcula los par�metros necesarios para recorrer la grilla
			calcularInicioGrilla(rayoactual.dir, puntoEntrada, incre, tMin);

			
			bool interseque = false;
			int menor=-1;
			float distancia;
	
			while(!salirGrilla){
				int indiceGrilla = (voxelActual.z * dimension_grilla.y* dimension_grilla.x) + (voxelActual.y * dimension_grilla.x) + voxelActual.x;
				int comienzoLista = tex1Dfetch(textura_voxels, indiceGrilla);
				menor= comienzoLista;
				bool intersecaron = false;
				distancia = configuracion_gpu.INFINITO;
				
				//Interseco con los objetos de la celda de la grilla
				if(comienzoLista!=-1){
					intersecaron = intersecarObjetosGrilla(comienzoLista, rayoactual, distancia, normal, menor, true);
					normal = normalize(normal);
					if(intersecaron){
						float3 prod = rayoactual.dir * distancia;
						float3 origen;
						origen = rayoactual.origen + prod;
						origen = coordMundoACoordGrid(origen);
						if(!((origen.x == voxelActual.x) && (origen.y == voxelActual.y) && (origen.z == voxelActual.z))){
							intersecaron = false;
						}
					}
				}

				if(intersecaron){					
					//Genero rayo a la luz
					Rayo rayoSombra;
					//CALCULO LA SOMBRA
					float sombra = 0.0;
					float3 origen = rayoactual.origen + (distancia * rayoactual.dir);
					
					float3 dirSombra;

					//TODO MAS LUCES
					//Para todas las luces
					int ind_luces = 0;
					int id_material = tex1Dfetch(textura_triangulos, 3 * menor).w;
					float3 colorDif = make_float3(tex1Dfetch(textura_materiales,4* id_material));
					float3 colorAmb = make_float3(tex1Dfetch(textura_materiales, 1+4*id_material));
					color = colorAmb;
		
					while((ind_luces < (int)cant_luces)){
						dirSombra = make_float3(tex1Dfetch(textura_luces, ind_luces*2)) - origen;
						rayoSombra.dir = normalize(dirSombra);
						rayoSombra.origen = origen + rayoSombra.dir * 1000000 * configuracion_gpu.ZERO;

						bool salir_sombra = false;
						float3 entradaSombra= make_float3(0,0,0);
						float3 increSombra= make_float3(0,0,0);
						float3 tMinSombra = make_float3(0,0,0);
						float3 normalS= make_float3(0,0,0);
						int menorS=0;					
						
						calcularInicioGrilla(rayoSombra.dir,rayoSombra.origen, increSombra, tMinSombra);

						entradaSombra = coordMundoACoordGrid(rayoSombra.origen);
						sombra = 0.f;
						
						while (sombra<1.0f && !salir_sombra){
							int indiceGrillaS = (entradaSombra.z * dimension_grilla.y* dimension_grilla.x) + (entradaSombra.y * dimension_grilla.x) + entradaSombra.x;

							int comienzoListaS = tex1Dfetch(textura_voxels, indiceGrillaS);
							float distanciaS = configuracion_gpu.INFINITO;						
							
							//Interseco con los objetos de la celda de la grilla
							if(comienzoListaS!=-1){
								bool mas_sombra = intersecarObjetosGrilla(comienzoListaS, rayoSombra, distanciaS, normalS, menorS, false);
								if(mas_sombra){
									//TODO calcular incremento de la sombra
									sombra=1.0;
								}
							}
							if(sombra<1.0){
								salir_sombra = !siguienteVoxel(tMinSombra, increSombra.x, increSombra.y, increSombra.z, entradaSombra);
							}
						}
						if( sombra < 1.0 )
						{
							float LxN = dot(rayoSombra.dir,normal);
							if (LxN>0)
							{
								float3 luz = make_float3(tex1Dfetch(textura_luces, ind_luces*2+1));
								color+= colorDif * luz * LxN;
							}
						}
						ind_luces++;
					}

					interseque = true;
				    salirGrilla = true;
				}
	
				if(!salirGrilla){
					if(!siguienteVoxel(tMin, incre.x, incre.y, incre.z, voxelActual)){
						salirGrilla = true;
						salir = true;
					}
				}
				
	
			}//WHILE NOT SALIR DE LA GRILLA
			
			if(interseque && nivel<configuracion_gpu.profundidad_recursion){
				int indice_material = (int)(tex1Dfetch(textura_triangulos, 3 * menor).w);
				float4 other = tex1Dfetch(textura_materiales, 3 + 4 * indice_material);
				float indiceEspecular = other.x + other.y;
				colorAcumulado = colorAcumulado + color * multiplicador * (1 - indiceEspecular);
				multiplicador *= indiceEspecular;
				if(other.x > 0){
					nivel++;
					
					float refraccion = adentro ? other.w : 1/other.w;

					float3 normal_refra = adentro ? -normal : normal;
					double cosI = -dot(normal_refra, rayoactual.dir);
					double cosT_cuadrado = 1.0f - refraccion * refraccion * (1.0f - cosI * cosI);

					if (cosT_cuadrado > 0) { //no ocurre la reflexi�n interna total
						 rayoactual.origen = (rayoactual.origen + (rayoactual.dir* distancia));
						 rayoactual.dir = normalize((rayoactual.dir * refraccion) + (normal_refra*(refraccion * cosI - sqrtf( cosT_cuadrado ))));
						 rayoactual.origen = rayoactual.origen + (rayoactual.dir * 10000 * configuracion_gpu.ZERO);
						 puntoEntrada = rayoactual.origen;
						 color = make_float3(0,0,0);
					}

					adentro = !adentro;

				}
				else if(other.y>0){
					nivel++;
					rayoactual.origen = (rayoactual.dir * (distancia)) + rayoactual.origen;
					//Calculo el rayo para la reflexion

					float VxN = dot(rayoactual.dir, normal) * 2.0f;
					rayoactual.dir = normalize(rayoactual.dir - normal * VxN);
					rayoactual.origen = rayoactual.origen+ normal * 100000*configuracion_gpu.ZERO;
					puntoEntrada = rayoactual.origen;
					color = make_float3(0,0,0);
				}
				else{
					salir = true;
				}
			}
			else{
				salir = true;//TODO???
				colorAcumulado = make_float3(0.5f,0.5f,1.0f);
			}
		}//WHILE not PROF && not SALIR
	
		if(!entra){
			colorAcumulado = make_float3(0.5f,0.5f,1.0f);
		}
		int indice = configuracion_gpu.resolucion.x*(blockIdx.y*blockDim.y+threadIdx.y)+(blockIdx.x*blockDim.x+threadIdx.x);
		retorno[indice] = colorAcumulado;
}


////<<<<<<<<<<<<< CALCULAR RAYOS >>>>>>>>>>>>>////
__global__ void calcularRayos(float4* rayos){
	int indiceRayo = configuracion_gpu.resolucion.x*(blockIdx.y*blockDim.y+threadIdx.y)+(blockIdx.x*blockDim.x+threadIdx.x);
	rayos[indiceRayo] = make_float4((ini - dy*(blockIdx.y*blockDim.y+threadIdx.y)+dx*(blockIdx.x*blockDim.x+threadIdx.x))-ojo,0);
	rayos[indiceRayo] = normalize(rayos[indiceRayo]);
}


/////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////
/////////////////    FUNCIONES PARA INVOCAR DE C   //////////////////////
/////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////

#ifdef CUDA_ENABLED

extern "C" void UpdateCamera(Escena* es, Configuracion conf){

	//SE ACTUALIZA LA CAMARA
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ojo),&(es->camara.ojo),sizeof(float3)));
	float3 dx_aca = make_float3((es->plano_de_vista.v2 - es->plano_de_vista.v1)/(float)conf.resolucion.x);
	float3 dy_aca = make_float3((es->plano_de_vista.v3 - es->plano_de_vista.v1)/(float)conf.resolucion.y);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dx),&(dx_aca),sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dy),&(dy_aca),sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ini),&(es->plano_de_vista.v3),sizeof(float3)));

	//SE RE-CALCULAN LOS RAYOS A TRAZAR
	dim3 gridR(conf.resolucion.x / conf.threads.x, conf.resolucion.y / conf.threads.y, 1);
	dim3 threadsR(conf.threads.x, conf.threads.y, 1);
	calcularRayos<<<gridR,threadsR>>>(d_listaRayos);

	//SE ACTUALIZA LA TEXTURA QUE CONTIENE LOS RAYOS
	textura_rayos.normalized = false;
	textura_rayos.filterMode = hipFilterModePoint;
	textura_rayos.addressMode[0]= hipAddressModeWrap;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	CUDA_SAFE_CALL(hipBindTexture(0, textura_rayos, d_listaRayos, channelDesc, conf.resolucion.x*conf.resolucion.y*sizeof(float4)));
}


extern "C" void initTexture(Escena* es, Configuracion conf){
	//SE COPIAN A CONSTANTES LOS VALORES
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(configuracion_gpu),&conf,sizeof(Configuracion)));
	float cantLucesCopy = static_cast<float>(es->cant_luces);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cant_luces),&(cantLucesCopy),sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(bounding_box),&(es->grilla.bbEscena),sizeof(BoundingBox)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dimension_grilla),&(es->grilla.dimension),sizeof(float3)));
	float3 tam_v = (es->grilla.bbEscena.maximum-es->grilla.bbEscena.minimum)/es->grilla.dimension;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(tam_voxel),&tam_v,sizeof(float3)));
	float3 tam_g = (es->grilla.bbEscena.maximum-es->grilla.bbEscena.minimum)/2.0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(tam_grilla),&tam_g,sizeof(float3)));

	//SE COPIAN LOS DATOS DE LOS TRIANGULOS A LA TEXTURA CORRESPONDIENTE
	float* d_triangulos;
	CUDA_SAFE_CALL( hipMalloc((void**) &d_triangulos, 3 * es->cant_objetos *sizeof(float4)));
	CUDA_SAFE_CALL(hipMemcpy(d_triangulos, es->triangulos, 3 * es->cant_objetos * sizeof(float4), hipMemcpyHostToDevice));
	textura_triangulos.normalized = false;
	textura_triangulos.filterMode = hipFilterModePoint;
	textura_triangulos.addressMode[0]= hipAddressModeWrap;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	size_t size = 3 * es->cant_objetos * sizeof(float4);
	CUDA_SAFE_CALL(hipBindTexture(0, textura_triangulos, d_triangulos, channelDesc, size));

	//SE COPIAN LOS DATOS DE LAS NORMALES A LA TEXTURA CORRESPONDIENTE
	float* d_normales;
	CUDA_SAFE_CALL( hipMalloc((void**) &d_normales, 3 * es->cant_objetos *sizeof(float4)));
	CUDA_SAFE_CALL(hipMemcpy(d_normales, es->normales, 3 * es->cant_objetos * sizeof(float4), hipMemcpyHostToDevice));
	textura_normales.normalized = false;
	textura_normales.filterMode = hipFilterModePoint;
	textura_normales.addressMode[0]= hipAddressModeWrap;
	CUDA_SAFE_CALL(hipBindTexture(0, textura_normales, d_normales, channelDesc, size));

	//SE COPIAN LOS DATOS DE LOS VOXELS A LA TEXTURA CORRESPONDIENTE
	hipChannelFormatDesc channelDescInt = hipCreateChannelDesc<int>();
	int cant_voxels = es->grilla.dimension.x * es->grilla.dimension.y * es->grilla.dimension.z;
	int* d_voxels;
	CUDA_SAFE_CALL( hipMalloc((void**) &d_voxels, cant_voxels *sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_voxels, es->grilla.voxels, cant_voxels * sizeof(int), hipMemcpyHostToDevice));
	textura_voxels.normalized = false;
	textura_voxels.filterMode = hipFilterModePoint;
	textura_voxels.addressMode[0]= hipAddressModeWrap;
	CUDA_SAFE_CALL(hipBindTexture(0, textura_voxels, d_voxels, channelDescInt, cant_voxels * sizeof(int)));
	
	//SE COPIAN LOS OBJETOS PERTENECIENTES A CADA VOXEL EN LA TEXTURA CORRESPONDIENTE
	int tam = -1;
	for(int i = 0; i<cant_voxels;i++){	
		tam = (tam>es->grilla.voxels[i])?tam:es->grilla.voxels[i];
	}
	while(es->grilla.listasGrid[tam]!=-1){
		tam++;
	}
	tam++;
	int* d_listasGrid;
	CUDA_SAFE_CALL( hipMalloc((void**) &d_listasGrid, tam*sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_listasGrid, es->grilla.listasGrid, tam* sizeof(int), hipMemcpyHostToDevice));
	textura_listasGrid.normalized = false;
	textura_listasGrid.filterMode = hipFilterModePoint;
	textura_listasGrid.addressMode[0]= hipAddressModeWrap;
	CUDA_SAFE_CALL(hipBindTexture(0, textura_listasGrid, d_listasGrid, channelDescInt, tam * sizeof(int)));

	//SE COPIAN LOS MATERIALES A LA TEXTURA CORRESPONDIENTE
	float* d_materiales;
	CUDA_SAFE_CALL( hipMalloc((void**) &d_materiales, es->cant_materiales*4*sizeof(float4)));
	CUDA_SAFE_CALL(hipMemcpy(d_materiales, (es->materiales), es->cant_materiales*4*sizeof(float4), hipMemcpyHostToDevice));
	textura_materiales.normalized = false;
	textura_materiales.filterMode = hipFilterModePoint;
	textura_materiales.addressMode[0]= hipAddressModeWrap;
	CUDA_SAFE_CALL(hipBindTexture(0, textura_materiales, d_materiales, channelDesc, es->cant_materiales*4*sizeof(float4)));

	//SE INICIALIZAN LA LISTA DE RAYOS Y LA MATRIZ EN LA QUE SE DEVOLVERA EL RESULTADO DEL RAYTRACE
	CUDA_SAFE_CALL( hipMalloc((void**) &d_color, sizeof(float3)* conf.resolucion.x * conf.resolucion.y));
	CUDA_SAFE_CALL( hipMalloc((void**) &d_listaRayos, conf.resolucion.x * conf.resolucion.y * sizeof(float4)));
}

extern "C" void initLuces(Escena* es, Configuracion conf){
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	//SE INICIALIZA LA TEXTURA DE LAS LUCES
	float* d_luces;
	CUDA_SAFE_CALL( hipMalloc((void**) &d_luces, 2 * es->cant_luces *sizeof(float4)));
	CUDA_SAFE_CALL(hipMemcpy(d_luces, es->luces, 2 * es->cant_luces * sizeof(float4), hipMemcpyHostToDevice));
	textura_luces.normalized = false;
	textura_luces.filterMode = hipFilterModePoint;
	textura_luces.addressMode[0]= hipAddressModeWrap;
	CUDA_SAFE_CALL(hipBindTexture(0, textura_luces, d_luces, channelDesc, 2 * es->cant_luces * sizeof(float4)));
}	


extern "C" void raytrace(Escena *es, Configuracion conf, float3 *imagen){
	int ancho = conf.resolucion.x;
	int alto = conf.resolucion.y;
	int threads_x = conf.threads.x;
	int threads_y = conf.threads.y;

	dim3 grid(ancho/threads_x, alto/threads_y, 1);
	dim3 threads(threads_x, threads_y, 1);
	
	hallarColor<<<grid,threads>>>(d_color);
	hipDeviceSynchronize();
	
	//COPIA DEL RESULTADO A LA IMAGEN.
	CUDA_SAFE_CALL( hipMemcpy(imagen, d_color, sizeof(float3) * ancho * alto , hipMemcpyDeviceToHost));
}


#endif

